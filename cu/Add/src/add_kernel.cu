#include "hip/hip_runtime.h"
#include <iostream>
#include "add.h"
//
// Created by sindre on 2023/1/28.
//

// ���������ӷ�kernel��grid��block��Ϊһά
__global__ void add(float* x, float * y, float* z, int n)
{
    // ��ȡȫ������
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // ����
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}


void get_cuda_info(){
    int dev=0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "ʹ��GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "�Դ��С��"  << devProp.totalGlobalMem / 1024.0 / 1024.0/ 1024.0<< " GB" << std::endl;
    std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
    std::cout << "ÿ���߳̿�Ĺ����ڴ��С��"  << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "ÿ��SM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;

}

void add_test()
{
    system("mode con cp select=936"); // 65001   UTF-8����ҳ  936��������Ĭ�ϵ�GBK
    get_cuda_info();
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // �����й��ڴ�
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // ��ʼ������
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // ����kernel��ִ������
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // ִ��kernel
    add << < gridSize, blockSize >> >(x, y, z, N);

    // ͬ��device ��֤�������ȷ����
    hipDeviceSynchronize();
    // ���ִ�н��
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "������: " << maxError << std::endl;

    // �ͷ��ڴ�
    hipFree(x);
    hipFree(y);
    hipFree(z);


    system("pause");
}